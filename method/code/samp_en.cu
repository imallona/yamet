#include "hip/hip_runtime.h"
#include <array>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#include "methData.h"
#include "samp_en.h"

__global__ void templateMatcher(int8_t *data, const unsigned int cumulativeSize,
                                const unsigned int m, unsigned int *d_prefixSum,
                                const unsigned int numBins, unsigned int *cm, unsigned int *cm_1) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= cumulativeSize - m || numBins == 0)
    return;
  unsigned int low  = 0;
  unsigned int high = numBins - 1;
  unsigned int mid;
  int          binIndex = -1;

  /// use binary search to find the bin to which the position belongs
  while (low <= high) {
    mid = low + (high - low) / 2;
    if (d_prefixSum[mid] == i) {
      binIndex = mid;
      break;
    } else if (d_prefixSum[mid] < i) {
      binIndex = mid;
      low      = mid + 1;
    } else {
      high = mid - 1;
    }
  }
  unsigned int binEnd  = (binIndex == numBins - 1) ? cumulativeSize : d_prefixSum[binIndex + 1];
  unsigned int binSize = binEnd - d_prefixSum[binIndex];
  unsigned int i_rel   = i - d_prefixSum[binIndex];

  if (i_rel >= binSize - m)
    return;

  bool         add = true;
  unsigned int idx = 0;
  for (unsigned int k = 0; k < m; k++) {
    if (data[i + k] == -1) {
      add = false;
      break;
    }
    idx += (data[i + k]) * (1 << k);
  }
  if (add && data[i + m] != -1) {
    atomicAdd(&cm[(1 << m) * binIndex + idx], 1);
    idx += (data[i + m]) * (1 << m);
    atomicAdd(&cm_1[(1 << (m + 1)) * binIndex + idx], 1);
  }
}

/**
 * Compute sample entropies at a bin level and aggregated at a file level for multiple files
 * simultaneously.
 *
 * @param fileMap FileMap object storing parsed methylation data of all files
 * @param m base length of templates
 * @return SampEns object storing per bin and file aggregated entropies
 */
SampEns sampEn(FileMap &fileMap, const unsigned int m, unsigned int n_streams) {
  SampEns sampens;
  Counts  counts;

  /**
   * initialise sampens with the default value of -1.0 which indicates that the sample entropy was
   * not computable
   * also initialise counts which keeps track of the total count of matching m-length and
   * (m+1)-length templates in every file
   */
  for (const auto &[file, fileMeths] : fileMap) {
    std::vector<std::vector<double>> x;
    for (const auto &chrBins : fileMeths)
      x.emplace_back(std::vector<double>(chrBins.meth.size(), -1.0));
    sampens[file] = FileSampEns{std::move(x), -1.0};
    counts[file]  = FileCounts{0, 0};
  }

  /// create CUDA streams
  hipStream_t streams[n_streams];
  for (unsigned short i = 0; i < n_streams; ++i) {
    hipStreamCreate(&streams[i]);
  }

  /**
   * array of pointers where each pointer is responsible for all methylation information for a
   * chromosome in a file
   */
  int8_t *d_flatBins[n_streams];
  /**
   * array of pointers where each pointer is responsible for tracking the number of m-length
   * templates of different types for a chromosome in a file
   */
  unsigned int *d_cm[n_streams];
  /**
   * array of pointers where each pointer is responsible for tracking the number of (m+1)-length
   * templates of different types for a chromosome in a file
   */
  unsigned int *d_cm_1[n_streams];
  /**
   * array of pointers where each pointer is responsible for tracking the starting positions of non
   * empty bins in d_flatBins
   */
  unsigned int *d_prefix_sum[n_streams];
  /**
   * array of pointers where each pointer is responsible for tracking the original indices of non
   * empty bins in a chromosome of a file
   */
  std::vector<unsigned int> goodBins[n_streams] = {};
  /**
   * array of pointers where each pointer is responsible for tracking the total number of positions
   * in a chromosome of a file
   */
  unsigned int cumulativeSize[n_streams] = {0};
  /**
   * array of pairs keeping track of the current filenames and chromosomes being worked on by the
   * different streams
   */
  std::pair<std::string, unsigned int> streamInfo[n_streams] = {{"", 0}};

  unsigned int   fileIndex = 0;
  unsigned short streamIdx = 0;

  for (const auto &[file, fileMeths] : fileMap) {
    for (unsigned int chrIndex = 0; chrIndex < fileMeths.size(); chrIndex++) {
      streamInfo[streamIdx] = {file, chrIndex};
      std::vector<unsigned int> prefix_sum;

      /// store the starting indices of good bins for a stream
      for (unsigned int i = 0; i < fileMeths[chrIndex].meth.size(); i++) {
        if (!fileMeths[chrIndex].meth[i].empty()) {
          goodBins[streamIdx].push_back(i);
          prefix_sum.push_back(cumulativeSize[streamIdx]);
          cumulativeSize[streamIdx] += fileMeths[chrIndex].meth[i].size();
        }
      }

      hipMallocAsync(&d_flatBins[streamIdx], cumulativeSize[streamIdx] * sizeof(int8_t),
                      streams[streamIdx]);
      hipMallocAsync(&d_prefix_sum[streamIdx], prefix_sum.size() * sizeof(unsigned int),
                      streams[streamIdx]);
      hipMallocAsync(&d_cm[streamIdx],
                      goodBins[streamIdx].size() * (1 << m) * sizeof(unsigned int),
                      streams[streamIdx]);
      hipMallocAsync(&d_cm_1[streamIdx],
                      goodBins[streamIdx].size() * (1 << (m + 1)) * sizeof(unsigned int),
                      streams[streamIdx]);

      /// store methylation information of all the bins in a flattened array
      for (unsigned int i = 0; i < goodBins[streamIdx].size(); i++) {
        unsigned int rowIndex = goodBins[streamIdx][i];
        hipMemcpyAsync(d_flatBins[streamIdx] + prefix_sum[i],
                        fileMeths[chrIndex].meth[rowIndex].data(),
                        fileMeths[chrIndex].meth[rowIndex].size() * sizeof(int8_t),
                        hipMemcpyHostToDevice, streams[streamIdx]);
      }

      hipMemcpyAsync(d_prefix_sum[streamIdx], prefix_sum.data(),
                      prefix_sum.size() * sizeof(unsigned int), hipMemcpyHostToDevice,
                      streams[streamIdx]);
      hipMemsetAsync(d_cm[streamIdx], 0,
                      goodBins[streamIdx].size() * (1 << m) * sizeof(unsigned int),
                      streams[streamIdx]);
      hipMemsetAsync(d_cm_1[streamIdx], 0,
                      goodBins[streamIdx].size() * (1 << (m + 1)) * sizeof(unsigned int),
                      streams[streamIdx]);

      unsigned int threadsPerBlock = 1024;
      unsigned int numBlocks = (cumulativeSize[streamIdx] + threadsPerBlock) / threadsPerBlock;

      templateMatcher<<<numBlocks, threadsPerBlock, 0, streams[streamIdx]>>>(
          d_flatBins[streamIdx], cumulativeSize[streamIdx], m, d_prefix_sum[streamIdx],
          goodBins[streamIdx].size(), d_cm[streamIdx], d_cm_1[streamIdx]);

      streamIdx++;

      if (streamIdx == n_streams ||
          (fileIndex == fileMap.size() - 1 && chrIndex == fileMeths.size() - 1)) {
        for (unsigned int j = 0; j < streamIdx; j++) {
          hipStreamSynchronize(streams[j]);
          unsigned int *h_cm   = new unsigned int[goodBins[j].size() * (1 << m)]();
          unsigned int *h_cm_1 = new unsigned int[goodBins[j].size() * (1 << (m + 1))]();

          hipMemcpy(h_cm, d_cm[j], goodBins[j].size() * (1 << m) * sizeof(unsigned int),
                     hipMemcpyDeviceToHost);
          hipMemcpy(h_cm_1, d_cm_1[j], goodBins[j].size() * (1 << (m + 1)) * sizeof(unsigned int),
                     hipMemcpyDeviceToHost);

          // std::cout << "file: " << streamInfo[j].first << std::endl;
          // std::cout << "  chr: " << fileMap[streamInfo[j].first][streamInfo[j].second].chr <<
          // std::endl;

          for (unsigned int i = 0; i < goodBins[j].size(); i++) {
            // std::cout << "    Bin " << i << ":" << std::endl;
            // std::cout << "      cm:" << h_cm[i] << std::endl;
            // std::cout << "      cm_1:" << h_cm_1[i] << std::endl;
            unsigned long long cm   = 0;
            unsigned long long cm_1 = 0;

            /// compute the number of matching m-length and (m+1)-length templates
            for (unsigned int k = 0; k < (1 << (m + 1)); k++) {
              if (k < (1 << m) && h_cm[(1 << m) * i + k] > 1) {
                cm += ((unsigned long long)h_cm[(1 << m) * i + k] *
                       (unsigned long long)(h_cm[(1 << m) * i + k] - 1)) /
                      2;
              }
              if (h_cm_1[(1 << (m + 1)) * i + k] > 1) {
                cm_1 += ((unsigned long long)(h_cm_1[(1 << (m + 1)) * i + k]) *
                         (unsigned long long)(h_cm_1[(1 << (m + 1)) * i + k] - 1)) /
                        2;
              }
            }
            /**
             * compute sample entropy only when the number of matching m-length and (m+1)-length
             * templates are non-zero
             */
            if (cm != 0 && cm_1 != 0) {
              sampens[streamInfo[j].first].raw[streamInfo[j].second][goodBins[j][i]] =
                  log((double)cm / (double)cm_1);
              counts[streamInfo[j].first].cm += cm;
              counts[streamInfo[j].first].cm_1 += cm_1;
            }
          }
          // std::cout << std::endl
          //           << std::endl;

          hipFreeAsync(d_flatBins[j], streams[j]);
          hipFreeAsync(d_prefix_sum[j], streams[j]);
          hipFreeAsync(d_cm[j], streams[j]);
          hipFreeAsync(d_cm_1[j], streams[j]);
          goodBins[j].clear();
          cumulativeSize[j] = 0;

          delete[] h_cm;
          delete[] h_cm_1;
        }
        streamIdx = 0;
      }
    }
    fileIndex++;
  }
  // std::cout << std::endl;

  /// compute aggregated sample entropies at file level
  for (auto &[file, samp] : sampens) {
    if (counts[file].cm > 0 && counts[file].cm_1 > 0) {
      samp.agg = log((double)counts[file].cm / (double)counts[file].cm_1);
    }
  }

  return sampens;
}

// for (unsigned chrIndex = 0; chrIndex < meths.size(); chrIndex++)
// {
// std::vector<unsigned int> prefixSum;
// std::vector<unsigned int> goodBins;
// unsigned int *cm;
// unsigned int *cm_1;
// unsigned int *d_prefixSum;
// unsigned int cumulativeSize = 0;

// for (unsigned int i = 0; i < meths[chrIndex].meth.size(); i++)
// {
//   if (!meths[chrIndex].meth[i].empty())
//   {
//     goodBins.push_back(i);
//     prefixSum.push_back(cumulativeSize);
//     cumulativeSize += meths[chrIndex].meth[i].size();
//   }
// }

// hipMalloc((void **)&d_flatBins, cumulativeSize * sizeof(char));

// for (unsigned int i = 0; i < goodBins.size(); i++)
// {
//   unsigned int rowIndex = goodBins[i];
//   hipMemcpy(d_flatBins + prefixSum[i], meths[chrIndex].meth[rowIndex].data(),
//              meths[chrIndex].meth[rowIndex].size() * sizeof(char), hipMemcpyHostToDevice);
// }

// hipMalloc(&d_prefixSum, prefixSum.size() * sizeof(unsigned int));
// hipMalloc((void **)&cm, goodBins.size() * sizeof(unsigned int));
// hipMalloc((void **)&cm_1, goodBins.size() * sizeof(unsigned int));

// hipMemset(cm, 0, goodBins.size() * sizeof(unsigned int));
// hipMemset(cm_1, 0, goodBins.size() * sizeof(unsigned int));

// hipMemcpy(d_prefixSum, prefixSum.data(), prefixSum.size() * sizeof(unsigned int),
// hipMemcpyHostToDevice);

// dim3 threadsPerBlock(32, 32);
// dim3 numBlocks((cumulativeSize + threadsPerBlock.x) / threadsPerBlock.x, (cumulativeSize +
// threadsPerBlock.y) / threadsPerBlock.y);

// templateMatcher<<<numBlocks, threadsPerBlock>>>(d_flatBins, cumulativeSize, m, d_prefixSum,
// goodBins.size(), cm, cm_1);

// unsigned int *h_cm = new unsigned int[goodBins.size()];
// unsigned int *h_cm_1 = new unsigned int[goodBins.size()];

// hipMemcpy(h_cm, cm, goodBins.size() * sizeof(unsigned int), hipMemcpyDeviceToHost);
// hipMemcpy(h_cm_1, cm_1, goodBins.size() * sizeof(unsigned int), hipMemcpyDeviceToHost);

// hipFree(d_flatBins);
// hipFree(d_prefixSum);
// hipFree(cm);
// hipFree(cm_1);

// std::cout << "chr: " << meths[chrIndex].chr << std::endl;
// std::cout << "  cm: ";
// for (unsigned int i = 0; i < goodBins.size(); i++)
// {
//   std::cout << h_cm[i] << " ";
// }
// std::cout << std::endl;

// std::cout << "  cm_1: ";
// for (unsigned int i = 0; i < goodBins.size(); i++)
// {
//   std::cout << h_cm_1[i] << " ";
//   if (h_cm[i] != 0 && h_cm_1[i] != 0)
//     sampens[chrIndex][goodBins[i]] = log((double)h_cm[i] / (double)h_cm_1[i]);
// }
// std::cout << std::endl;
// }
// return sampens;

// DataRow *d_data;
// unsigned int *d_cm, *d_cm_1;
// unsigned int h_cm = 0, h_cm_1 = 0;

// hipMalloc(&d_data, N * sizeof(DataRow));
// hipMalloc(&d_cm, sizeof(unsigned int));
// hipMalloc(&d_cm_1, sizeof(unsigned int));

// hipMemcpy(d_data, data.data(), N * sizeof(DataRow), hipMemcpyHostToDevice);
// hipMemcpy(d_cm, &h_cm, sizeof(unsigned int), hipMemcpyHostToDevice);
// hipMemcpy(d_cm_1, &h_cm_1, sizeof(unsigned int), hipMemcpyHostToDevice);

// dim3 threadsPerBlock(32, 32);
// dim3 numBlocks((N - m + threadsPerBlock.x) / threadsPerBlock.x, (N - m + threadsPerBlock.y) /
// threadsPerBlock.y);

// hipEvent_t start, stop;
// hipEventCreate(&start);
// hipEventCreate(&stop);
// hipEventRecord(start);
// templateMatcher<<<numBlocks, threadsPerBlock>>>(d_data, N, m, d_cm, d_cm_1);
// hipEventRecord(stop);
// hipEventSynchronize(stop);

// size_t free_mem, total_mem;
// hipMemGetInfo(&free_mem, &total_mem);
// std::cout << "GPU Memory Used: " << (total_mem - free_mem) / (1024 * 1024) << "MB" <<
// std::endl;

// float milliseconds = 0;
// hipEventElapsedTime(&milliseconds, start, stop);
// std::cout << "Sample Entropy computation time: " << milliseconds << " ms" << std::endl;

// hipEventDestroy(start);
// hipEventDestroy(stop);

// hipMemcpy(&h_cm, d_cm, sizeof(unsigned int), hipMemcpyDeviceToHost);
// hipMemcpy(&h_cm_1, d_cm_1, sizeof(unsigned int), hipMemcpyDeviceToHost);

// hipFree(d_data);
// hipFree(d_cm);
// hipFree(d_cm_1);
// return log((double)h_cm / (double)h_cm_1);

// CPU only code below

// double sampEn(std::vector<DataRow> &data, const int m)
// {
//   unsigned int N = data.size();
//   unsigned int cm = 0, cm_1 = 0;
//   bool eq = true;
//   for (unsigned int i = 0; i < N - m; i++)
//   {
//     for (unsigned int j = i + 1; j < N - m; j++)
//     {
//       eq = true;
//       for (unsigned int k = 0; k < m; k++)
//       {
//         if (data[i + k].rate != data[j + k].rate)
//         {
//           eq = false;
//           break;
//         }
//       }
//       if (eq)
//         cm++;
//       if (eq && data[i + m].rate == data[j + m].rate)
//         cm_1++;
//     }
//   }
//   for (unsigned int i = 0; i < N - m; i++)
//   {
//     eq = true;
//     for (unsigned int k = 0; k < m; k++)
//     {
//       if (data[i + k].rate != data[N - m + k].rate)
//       {
//         eq = false;
//         break;
//       }
//     }
//     if (eq)
//       cm++;
//   }
//   return log((double)cm / (double)cm_1);
// }

// a transformation like
// unsigned int j = 1 + static_cast<unsigned int>((-1 + std::sqrt(8.0 * tid + 1)) / 2);
// unsigned int i = tid - (j * (j - 1)) / 2;
// could improve computational speed by considering the upper diagonal structure